#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb.h>
#include <optixu/optixu_aabb_namespace.h>
#include "random.h"

using namespace optix;

rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(float, fov, , );

rtDeclareVariable(float4, diffuse, , );
rtDeclareVariable(int, texCount, , );

rtDeclareVariable(float4, lightDir, , );
rtDeclareVariable(float4, lightPos, , );
rtDeclareVariable(uint, frameCount, , );

rtDeclareVariable(rtObject, top_object, , );

rtBuffer<float4> vertex_buffer;
rtBuffer<uint> index_buffer;
rtBuffer<float4> normal;
rtBuffer<float4> texCoord0;
rtTextureSampler<float4, 2> tex0;

rtBuffer<float4, 2> output0;

struct PerRayDataResult {
    float4 result;
    int depth;
    uint seed;
    float entrance;
};

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );
rtDeclareVariable(PerRayDataResult, payload, rtPayload, );

rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(float3, texCoord, attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

rtDeclareVariable(int, Phong, , );
rtDeclareVariable(int, Shadow, , );

__device__ float FresnelPower = 0.9f;
__device__ float F = 0.039962f;

__device__ float2 lightSize = {0.4, 0.45};

#include "util.h"

RT_PROGRAM void raygen() {
    size_t2 screen = output0.size();
    unsigned int seed = tea<16>(screen.x * launch_index.y + launch_index.x, frameCount);

    float2 d = make_float2(launch_index) / make_float2(launch_dim) * 2.0f - 1.0f;
    float3 ray_origin = eye;
    float3 ray_direction = normalize(d.x*U*fov + d.y*V*fov + W);

    optix::Ray r = optix::make_Ray(ray_origin, ray_direction, Phong, 0.00000000001, RT_DEFAULT_MAX);

    PerRayDataResult rp;
    rp.result = make_float4(1.0f, 1.0f, 1.0f, 1.0f);
    rp.depth = 1;
    rp.seed = seed;
    rp.entrance = 0.0f;

    rtTrace(top_object, r, rp);

    output0[launch_index] = rp.result;
}


RT_PROGRAM void raygen_ms() {
    float4 color = make_float4(0.0);
	int sqrt_num_samples = 2;
	int samples = sqrt_num_samples * sqrt_num_samples;

	size_t2 screen = output0.size();

	float2 inv_screen = 1.0f/make_float2(screen) * 2.f;
	float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;

	float2 jitter_scale = inv_screen / sqrt_num_samples;
	float2 d = make_float2(launch_index) / make_float2(launch_dim) * 2.f - 1.f;

	float2 scale = 1 / (make_float2(launch_dim) * sqrt_num_samples) * 2.0f;
	unsigned int seed = tea<16>(screen.x*launch_index.y+launch_index.x, frameCount);

	for (int i = 0; i < sqrt_num_samples; ++i) {
		for (int j = 0; j < sqrt_num_samples; ++j) {

			float2 jitter = make_float2((i+1)+rnd(seed), (j+1)+rnd(seed));
			float2 d = pixel + jitter*jitter_scale;
			float3 ray_origin = eye;
			float3 ray_direction = normalize(d.x*U*fov + d.y*V*fov + W);
			
			optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, Phong, 0.000000001, RT_DEFAULT_MAX);
	
			PerRayDataResult prd;
			prd.result = make_float4(1.0f);
			prd.depth = 0;
			prd.seed = seed;
	
			rtTrace(top_object, ray, prd);
			color += prd.result;
		}
	}
	output0[launch_index] = color/samples;
}


RT_PROGRAM void exception() {
    output0[launch_index] = make_float4(1.0f, 0.0f, 0.0f, 1.0f);
}


RT_PROGRAM void tracePath() {
    if (payload.depth >= 4) return;
    
    float3 n = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    float3 hit_point = ray.origin + t_hit * ray.direction;
    
    float4 shadow = sampleAreaLight(n, hit_point, make_float3(lightPos.x, lightPos.y, lightPos.z), make_float3(0, -1, 0), lightSize.x, lightSize.y, payload.seed);
    float4 color = diffuse;
    payload.result *= color * shadow;
}


RT_PROGRAM void miss() {
    payload.result = make_float4(1.0f, 1.0f, 1.0f, 1.0f);
}


RT_PROGRAM void missShadow() {
    payload.result = make_float4(1.0f, 1.0f, 1.0f, 1.0f);
}


RT_PROGRAM void any_hit_shadow() {
    payload.result = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
}


RT_PROGRAM void traceGlass() {
    if (payload.depth >= 4) return;

    float3 hit_point = ray.origin + t_hit * ray.direction;
    float3 n = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));

    float fresnel = dot(-normalize(ray.direction), n);
    float ratio = F + (1.0f - F) * pow((1.0f - fresnel), FresnelPower);

    float3 reflectDir = reflect(ray.direction, n);
    float3 refractDir = ray.direction;
    
    optix::Ray refl_ray(hit_point, reflectDir, Phong, 0.001, 5000);
    optix::Ray refr_ray(hit_point, refractDir, Phong, 0.001, 5000);

    PerRayDataResult reflPayload = payload;
    PerRayDataResult refrPayload = payload;

    rtTrace(top_object, refl_ray, reflPayload);
    rtTrace(top_object, refr_ray, refrPayload);

    //payload.result = make_float4((lerp(refrPayload.result, reflPayload.result, ratio) * 0.9f).rgb, 1);
    float4 tmp = lerp(refrPayload.result, reflPayload.result, ratio) * 0.9f;
    payload.result = make_float4(tmp.x, tmp.y, tmp.z, 1.0f);
}


RT_PROGRAM void keepGoingShadow() {

    float3 n = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    float atenuation = 1.0f;
    atenuation *= sqrt(abs(dot(n, ray.direction)));

    payload.result *= atenuation;

    rtIgnoreIntersection();
}


RT_PROGRAM void geometryintersection(int primIdx) {

	float4 vecauxa = vertex_buffer[index_buffer[primIdx*3]];
	float4 vecauxb = vertex_buffer[index_buffer[primIdx*3+1]];
	float4 vecauxc = vertex_buffer[index_buffer[primIdx*3+2]];

	float3 v0 = make_float3(vecauxa);
	float3 v1 = make_float3(vecauxb);
	float3 v2 = make_float3(vecauxc);

    float3 n;
    float  t, beta, gamma;
    if( intersect_triangle( ray, v0, v1, v2, n, t, beta, gamma ) ) {

        if(  rtPotentialIntersection( t ) ) {

            float3 n0 = make_float3(normal[ index_buffer[primIdx*3]]);
            float3 n1 = make_float3(normal[ index_buffer[primIdx*3+1]]);
            float3 n2 = make_float3(normal[ index_buffer[primIdx*3+2]]);

            float3 t0 = make_float3(texCoord0[ index_buffer[primIdx*3]]);
            float3 t1 = make_float3(texCoord0[ index_buffer[primIdx*3+1]]);
            float3 t2 = make_float3(texCoord0[ index_buffer[primIdx*3+2]]);

            shading_normal   = normalize( n0*(1.0f-beta-gamma) + n1*beta + n2*gamma );
            texCoord =  t0*(1.0f-beta-gamma) + t1*beta + t2*gamma ;
            geometric_normal = normalize( n );

            rtReportIntersection(0);
        }
    }
}


RT_PROGRAM void boundingbox(int primIdx, float result[6]) {

	float3 v0 = make_float3(vertex_buffer[index_buffer[primIdx*3]]);
	float3 v1 = make_float3(vertex_buffer[index_buffer[primIdx*3+1]]);
	float3 v2 = make_float3(vertex_buffer[index_buffer[primIdx*3+2]]);  
	
	const float  area = length(cross(v1-v0, v2-v0));

	optix::Aabb* aabb = (optix::Aabb*)result;

	if(area > 0.0f && !isinf(area)) {
		aabb->m_min = fminf( fminf( v0, v1), v2 );
		aabb->m_max = fmaxf( fmaxf( v0, v1), v2 );
	} 
	else {
	    aabb->invalidate();
	}
}